#include "hip/hip_runtime.h"
#include <stdio.h>
#include <sys/time.h>
#include <cmath>
#include "hip/hip_runtime.h"

#include "toolbox.hpp"


template<class T>
void sinus_cpu(const T* input, T* output, const size_t size) {
    for(int i=0;i<size;++i) {
        output[i] = sin(input[i]);
    }
}

template<class T>
void sinus_par(const T* input, T* output, const size_t size) {
#pragma omp parallel for
    for(int i=0;i<size;++i) {
        output[i] = sin(input[i]);
    }
}

template<class T>
__global__ void sinus(const T* input, T* output, const uint size) {
    const int tidx = threadIdx.x + blockDim.x * blockIdx.x;
    if(tidx<size)
        output[tidx] = sin(input[tidx]);
}
template<class T>
__global__ void sinus(const T* input, T* output, const uint size, const uint iterations) {
    /*
    const int tidx = threadIdx.x + blockDim.x * blockIdx.x;
    for(int i = 0;i<iterations;++i)
        output[idx + blockDim.x * blockIdx.x] = sin();
    */
}

struct sin_time {
    double cpu;
    double gpu;
    double par_gpu;
};

template<class T>
void test_sinus(const size_t input_size, const uint wg_size, timestruct& tms) {
    double t1,t2;
    T *dev_a, *dev_b;
    T *h_a,*h_b;

    /**init**/

    h_a = (T*)malloc(input_size*sizeof(T));
    h_b = (T*)malloc(input_size*sizeof(T));

    hipMalloc( (void**)&dev_a, input_size*sizeof(T));
    hipMalloc( (void**)&dev_b, input_size*sizeof(T));

#pragma omp parallel for
    for(int i = 0;i<input_size;++i) {
        h_a[i]=i;
    }
    hipMemcpy(dev_a,h_a,input_size*sizeof(T),hipMemcpyHostToDevice);

    //GPU
    {
        const int nblocks = input_size / wg_size +1;

        t1 = time_in_seconds();
        sinus<<<nblocks,wg_size>>>(dev_a,dev_b,input_size);
        hipDeviceSynchronize();
        t2 = time_in_seconds();
        printf("Time Sinus GPU: %.3fs\n", t2-t1);
        tms.add(t2-t1,0,"Sinus GPU");
    }
    //CPU
    {
        t1 = time_in_seconds();
        sinus_cpu(h_a,h_b,input_size);
        t2 = time_in_seconds();
        printf("Time Sinus CPU: %.3fs\n", t2-t1);
        tms.add(t2-t1,1,"Sinus CPU");
    }
    //CPU
    {
        t1 = time_in_seconds();
        sinus_par(h_a,h_b,input_size);
        t2 = time_in_seconds();
        printf("Time Sinus Par: %.3fs\n", t2-t1);
        tms.add(t2-t1,2,"Sinus Parallel CPU");
    }

    /***copy dev_b to h_a for checking, which is not needed anymore***/
    hipMemcpy(h_a,dev_b,input_size*sizeof(T),hipMemcpyDeviceToHost);
    for(int i = 0;i<input_size;++i) {
        if(!compare_float(h_b[i],h_a[i])) {
            fprintf(stderr,"Error at Position %d: %.3f (GPU) <> %.3f (CPU)\n",i,h_a[i],h_b[i]);
            break;
        }
    }
    free(h_a);
    free(h_b);
    hipFree(dev_a);
    hipFree(dev_b);
}

int main(int argc, char** argv) {
    uint s = 1000*1000*10;
    uint wg_size = 256;

    if(argc>1) {
        s = atoi(argv[1]);
    }
    printf("Size: %d\n",s);
    if(argc>2) {
        wg_size = atoi(argv[2]);
        printf("Workgroup Size: %d\n", wg_size);
    }

    timestruct tms_double;
    timestruct tms_float;

    for(int i=0;i<5;++i) {
        printf("=== Double ===\n");
        test_sinus<double>(s, wg_size,tms_double);

        printf("=== Float ===\n");
        test_sinus<float>(s, wg_size,tms_float);
    }
    printf("#double;%d;",s);
    tms_double.print();
    printf("#float;%d;",s);
    tms_float.print();
}
